/*
 * @Author: liaoyixiong 
 * @Date: 2021-12-25 17:48:34 
 * @Last Modified by:   liaoyixiong 
 * @Last Modified time: 2021-12-25 17:48:34 
 */

#include "stdio.h"
#include <hip/hip_runtime.h>



/*
cuda核函数
__global__ 核函数的前缀定义
   - 使用__global__修饰的函数，必须是void无返回值
   - __global__核函数修饰，必须是nvcc编译才有效，否则无效
   - __global__修饰的函数， 使用name<<<grid, block, memory, stream>>>(params)启动核函数 
         - 启动在host, 但执行在device
定义如下：
__device__, 函数执行在设备上
__global__, 函数执行在设备上，但是调用在host上，定义核函数的符号
__host__, 函数执行在host上，调用也在host上

__device__ 修饰的函数， 只能在设备上执行，设备上调用(例如核函数内调用)
    - nvidia提供了很多内置设备函数，比如日常的cos, sin之类的
        - 在nvidia团队中， 不同的内置函数的api接口版本号，被称为计算能力

*/




// sigmoid 不能够使用 sigmoid<<<1, 3>>>这种启动它
// 也不能直接 sigmoid(0.1)
// 只能在核函数内调用他

__device__ float sigmoid(float value) {
  return 1 / (1 + exp(-value));
}





__global__ void compute(float* a, float* b, float* c) {

  /*  
    线程layout的概念， 启动文档线程会被设计为gird和block, 如同提高的参数一样
    这个layohut的概念是虚拟的，通过cuda驱动实现真实硬件映射，抽象了一个中间值（调度层）
      - 如果我们有4353 Core
      - 如果我们需要启动5000个线程
      - 抽象层它会把5000个线程安排到各个Core中执行， 根据情况来执行次数
      - 每次调度单位为WarpSize, 如果启动的线程不足，也会执行WarpSize, 不过Core是非激活状态而已
    
    需要启动多少个线程，通过girdDim和blockDim告诉它
    线程数= gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z 


    2147483647  是有符号整数(int)的最大值
    65535       是无符号整数(int16m short)的最大值
    gridDim的最大值范围 ： (x,y,z): (2147483647, 65535, 65535)
    blockDim的最大值范围： (x,y,z): (1024, 1024, 64)



    它的定义在device_launch_parameters.h
    uint3 __device_builtin__ __STORAGE__ threadIdx;
    uint3 __device_builtin__ __STORAGE__ blockIdx;
    dim3 __device_builtin__ __STORAGE__ blockDim;
    dim3 __device_builtin__ __STORAGE__ gridDim;
    int __device_builtin__ __STORAGE__ warpSize;

    获取线程ID, 进行数据操作
    数据索引， 是通过blockIdx和threadIdx计算得到
    girdDim告诉你Grid的大小， blockDim告诉你block大小
    blockIdx告诉你所在Grid内的索引， threadIdx告诉你所在block内的索引
    把gridDim和blockDim设想为一个tensor
    则：
    gridDim的shape  = gridDim.z * gridDim.y * gridDim.x
    blockDim的shape = blockDim.z * blockDim.y * blockDim.x
    最终的启动线程的shape维度为  gridDim.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x

    如果启动的线程是6个维度的tensor, 那么索引，也可以类似
    blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x

    仅仅是在这个场景需要把6个维度索引变为连续的内存单元索引
    如果有6个维度a, b, c, d, e, f和6个位置的索引u, v, w, x, y, z
    a     u
    b     v  
    c     w
    d     x  
    e     y
    f     z  
    position = ((((u * b + v) * c + w) * d + x) * e + y) * f + z

  */

  // gridDim  = 1 * 1 * 1
  // blockDim = 3 * 1 * 1 

  int position = blockDim.x * blockIdx.x + threadIdx.x;
  c[position] = a[position] * sigmoid(b[position]);
}


int main() {

  const int num = 3;
  float a[num] = {1, 2, 3};
  float b[num] = {5, 7, 9};
  float c[num] = {0};
  
  size_t size_array = sizeof(c);
  float* device_a = nullptr;
  float* device_b = nullptr;
  float* device_c = nullptr;
  
  hipMalloc(&device_a, size_array);
  hipMalloc(&device_b, size_array);
  hipMalloc(&device_c, size_array);

  hipMemcpy(device_a, a, size_array, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b, size_array, hipMemcpyHostToDevice);

  compute<<<dim3(1), dim3(3)>>>(device_a, device_b, device_c);

  hipMemcpy(c, device_c, size_array, hipMemcpyDeviceToHost);

  for (int i = 0; i < num; ++i) {
    printf("c[%d] = %f\n", i, c[i]);
  }
  return 0;






}